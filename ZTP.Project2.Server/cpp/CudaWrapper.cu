#include "hip/hip_runtime.h"
﻿#include "CUDAWrapper.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void grayscaleKernel(unsigned char* input, unsigned char* output, int pixelCount) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < pixelCount) {
        int idx = i * 3;
        unsigned char r = input[idx];
        unsigned char g = input[idx + 1];
        unsigned char b = input[idx + 2];
        unsigned char gray = static_cast<unsigned char>((r + g + b) / 3);
        output[idx] = gray;
        output[idx + 1] = gray;
        output[idx + 2] = gray;
    }
}

void ProcessGrayscale(unsigned char* input, int length, unsigned char* output) {
    int totalPixels = length / 3;
    int totalBytes = length;

    unsigned char* devInput;
    unsigned char* devOutput;

    hipMalloc((void**)&devInput, totalBytes);
    hipMalloc((void**)&devOutput, totalBytes);

    hipMemcpy(devInput, input, totalBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (totalPixels + threadsPerBlock - 1) / threadsPerBlock;

    grayscaleKernel<<<blocks, threadsPerBlock>>>(devInput, devOutput, totalPixels);

    hipMemcpy(output, devOutput, totalBytes, hipMemcpyDeviceToHost);

    hipFree(devInput);
    hipFree(devOutput);
}
